#include "hip/hip_runtime.h"
// this source is derived from CHILL AST originally from file 'GPU/mm/mm.c' as parsed by frontend compiler rose

__global__ void kernel_gpu(double *c[3968], double *b[3968], double *a[3968]) {
  __shared__ double _P2[64][17];
  __shared__ double _P1[16][65];
  double _P3[4][4];
  int k;
  int kk;
  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int by = blockIdx.y;
  int bx = blockIdx.x;
  {
    {
      {
        {
          _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] = c[64 * by + ty][64 * bx + tx];
          _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] = c[64 * by + ty + 16][64 * bx + tx];
          _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] = c[64 * by + ty + 32][64 * bx + tx];
          _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] = c[64 * by + ty + 48][64 * bx + tx];
          _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] = c[64 * by + ty][64 * bx + tx + 16];
          _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] = c[64 * by + ty + 16][64 * bx + tx + 16];
          _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] = c[64 * by + ty + 32][64 * bx + tx + 16];
          _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] = c[64 * by + ty + 48][64 * bx + tx + 16];
          _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] = c[64 * by + ty][64 * bx + tx + 32];
          _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] = c[64 * by + ty + 16][64 * bx + tx + 32];
          _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] = c[64 * by + ty + 32][64 * bx + tx + 32];
          _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] = c[64 * by + ty + 48][64 * bx + tx + 32];
          _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] = c[64 * by + ty][64 * bx + tx + 48];
          _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] = c[64 * by + ty + 16][64 * bx + tx + 48];
          _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] = c[64 * by + ty + 32][64 * bx + tx + 48];
          _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] = c[64 * by + ty + 48][64 * bx + tx + 48];
        }
      }
      for (kk = 0; kk <= 3952; kk += 16) {
        {
          {
            _P1[tx + kk - kk][64 * by + ty - 64 * by] = b[64 * by + ty][tx + kk];
          }
        }
        __syncthreads();
        {
          {
            _P2[64 * bx + tx - 64 * bx][ty + kk - kk] = a[ty + kk][64 * bx + tx];
          }
        }
        __syncthreads();
        for (k = 0; k <= 15; k += 1) {
          {
            {
              _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] = _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] + _P2[64 * bx + tx - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty - 64 * by];
              _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] + _P2[64 * bx + tx - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 16 - 64 * by];
              _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] + _P2[64 * bx + tx - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 32 - 64 * by];
              _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16] + _P2[64 * bx + tx - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 48 - 64 * by];
              _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 16 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty - 64 * by];
              _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 16 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 16 - 64 * by];
              _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 16 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 32 - 64 * by];
              _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 16 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 48 - 64 * by];
              _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 32 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty - 64 * by];
              _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 32 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 16 - 64 * by];
              _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 32 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 32 - 64 * by];
              _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 32 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 48 - 64 * by];
              _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 48 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty - 64 * by];
              _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 48 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 16 - 64 * by];
              _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 48 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 32 - 64 * by];
              _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] = _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16] + _P2[64 * bx + tx + 48 - 64 * bx][k + kk - kk] * _P1[k + kk - kk][64 * by + ty + 48 - 64 * by];
            }
          }
          __syncthreads();
        }
        __syncthreads();
      }
      {
        {
          c[64 * by + ty][64 * bx + tx] = _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16];
          c[64 * by + ty + 16][64 * bx + tx] = _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16];
          c[64 * by + ty + 32][64 * bx + tx] = _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16];
          c[64 * by + ty + 48][64 * bx + tx] = _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx - (64 * bx + tx)) / 16];
          c[64 * by + ty][64 * bx + tx + 16] = _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16];
          c[64 * by + ty + 16][64 * bx + tx + 16] = _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16];
          c[64 * by + ty + 32][64 * bx + tx + 16] = _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16];
          c[64 * by + ty + 48][64 * bx + tx + 16] = _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 16 - (64 * bx + tx)) / 16];
          c[64 * by + ty][64 * bx + tx + 32] = _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16];
          c[64 * by + ty + 16][64 * bx + tx + 32] = _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16];
          c[64 * by + ty + 32][64 * bx + tx + 32] = _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16];
          c[64 * by + ty + 48][64 * bx + tx + 32] = _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 32 - (64 * bx + tx)) / 16];
          c[64 * by + ty][64 * bx + tx + 48] = _P3[(64 * by + ty - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16];
          c[64 * by + ty + 16][64 * bx + tx + 48] = _P3[(64 * by + ty + 16 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16];
          c[64 * by + ty + 32][64 * bx + tx + 48] = _P3[(64 * by + ty + 32 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16];
          c[64 * by + ty + 48][64 * bx + tx + 48] = _P3[(64 * by + ty + 48 - (64 * by + ty)) / 16][(64 * bx + tx + 48 - (64 * bx + tx)) / 16];
        }
      }
    }
  }
}
#define N 3968

void normalMM(double c[3968][3968], double a[3968][3968], double b[3968][3968]) {
  double * devRO1ptr;
  double * devRO0ptr;
  double * devRW3ptr;
  hipMalloc((void **)&devRW3ptr, 15745024 * sizeof(double));
  hipMemcpy(devRW3ptr, c, 15745024 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void **)&devRO0ptr, 15745024 * sizeof(double));
  hipMemcpy(devRO0ptr, a, 15745024 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void **)&devRO1ptr, 15745024 * sizeof(double));
  hipMemcpy(devRO1ptr, b, 15745024 * sizeof(double), hipMemcpyHostToDevice);
  dim3 dimGrid0 = dim3(62, 62);
  dim3 dimBlock0 = dim3(16, 16);
  kernel_gpu<<<dimGrid0,dimBlock0>>>((double (*)[3968])devRW3ptr, (double (*)[3968])devRO0ptr, (double (*)[3968])devRO1ptr);
  hipMemcpy(c, devRW3ptr, 15745024 * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(devRW3ptr);
  hipFree(devRO0ptr);
  hipFree(devRO1ptr);
}
